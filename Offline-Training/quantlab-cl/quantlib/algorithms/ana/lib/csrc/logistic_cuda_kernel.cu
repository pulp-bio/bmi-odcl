#include "hip/hip_runtime.h"
/*  
 *  logistic_cuda_kernel.cu
 *  
 *  Author(s):
 *  Matteo Spallanzani <spmatteo@iis.ee.ethz.ch>
 *  
 *  Copyright (c) 2020-2021 ETH Zurich.
 *  
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *  
 *  http://www.apache.org/licenses/LICENSE-2.0
 *  
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */ 

#include <torch/extension.h>
#include <vector>

// #include <stdio.h>  // for debug

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "forward.h"


#define THREADS_PER_BLOCK 1024


// definitions of CUDA kernels (executed on: GPU)

template <typename scalar_t>
__global__ void logistic_forward_pmf_cuda_kernel(
    scalar_t * const __restrict__ pmf,
    const scalar_t * __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ t,
    const int64_t len_t,
    const scalar_t * __restrict__ mi,
    const scalar_t * __restrict__ sigma,
    const scalar_t * __restrict__ training
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < len_x)
    {
        // pre-compute row offset from the beginning of the `pmf` array
        int row_offset = ix * PLUS_1(len_t);

        // compute shifted thresholds
        for (int it = 0; it < len_t; ++it)
        {
            pmf[row_offset + it + 1] = x_in[ix] - *mi - t[it];
        }

        // compute CDF
        for (int it = 0; it < PLUS_1(len_t); ++it)
        {
            if (it == 0)
            {
                pmf[row_offset + it] = 1.0f;
            }
            else
            {
                if (*training && (*sigma != 0.0f))
                {
                    scalar_t sigma_inv = 1.0f / (*sigma);
                    scalar_t shifted_x_minus_t_over_s = pmf[row_offset + it] * sigma_inv;
                    scalar_t exp_shifted_x_minus_t_over_s = expf(-1.0f * shifted_x_minus_t_over_s);
                    pmf[row_offset + it] = 1.0f / (1.0f + exp_shifted_x_minus_t_over_s);
                }
                else
                {
                    pmf[row_offset + it] = (scalar_t) (pmf[row_offset + it] >= 0.0f);
                }
            }
        }

        // compute the probability mass in each bin
        for (int iq = 0; iq < PLUS_1(len_t) - 1; ++iq)
        {
            pmf[row_offset + iq] = pmf[row_offset + iq] - pmf[row_offset + iq + 1];
        }
        // the last bin (with index `row_offset + len_t`) would have mass `pmf[row_offset + len_t] - 0.0f`, so it's not necessary to compute it!
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void logistic_backward_cuda_kernel(
    scalar_t * const __restrict__ grad_out,
    const scalar_t * __restrict__ grad_in,
    const scalar_t * __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ q,
    const scalar_t * __restrict__ t,
    const int64_t len_t,
    const scalar_t * __restrict__ mi,
    const scalar_t * __restrict__ sigma
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < len_x)
    {
        scalar_t sum = 0.0f;

        for (int it = 0; it < len_t; ++it)
        {
            // input position relative to the threshold
            scalar_t shifted_x_minus_t  = x_in[ix] - *mi - t[it];

            // the derivative of the expected (i.e., regularised) step function is the PDF of the logistic distribution
            scalar_t pdf;
            if (*sigma != 0.0f)
            {
                scalar_t sigma_inv = 1.0f / (*sigma);
                scalar_t shifted_x_minus_t_over_s = shifted_x_minus_t * sigma_inv;
                scalar_t exp_shifted_x_minus_t_over_s = expf(-1.0f * shifted_x_minus_t_over_s);
                scalar_t cdf = 1.0f / (1.0f + exp_shifted_x_minus_t_over_s);
                pdf = cdf * (1.0f - cdf) * sigma_inv;
            }
            else
            {
                pdf = 0.0f;  // no noise, no gradient!
            }

            // dilate and accumulate expected derivative
            scalar_t dq = q[it + 1] - q[it];
            sum += dq * pdf;
        }

        // compose gradients
        grad_out[ix] = sum * grad_in[ix];
    }
    else  // I am out of bounds!
    {
        return;
    }
}


// definitions of C++\CUDA interface (executed on: CPU)
// goals:
//   * allocate GPU memory for the output;
//   * define the parameters for the GPU kernel;
//   * call the kernel;

torch::Tensor logistic_forward_cuda_dispatch(
    torch::Tensor x_in,
    torch::Tensor q,
    torch::Tensor t,
    torch::Tensor mi,
    torch::Tensor sigma,
    torch::Tensor strategy,
    torch::Tensor training
)
{
    auto x_out = torch::zeros_like(x_in);
    auto pmf = torch::zeros({x_in.numel(), PLUS_1(t.numel())}, torch::TensorOptions().dtype(x_in.dtype()).device(x_in.device()));

    const dim3 blocks((x_in.numel() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    // compute PMF over bins (i.e., the quantization levels)
    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "logistic_forward_pmf_cuda_kernel",
        ([&] {
            logistic_forward_pmf_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                pmf.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                x_in.numel(),
                t.data_ptr<scalar_t>(),
                t.numel(),
                mi.data_ptr<scalar_t>(),
                sigma.data_ptr<scalar_t>(),
                training.data_ptr<scalar_t>()
            );
        })
    );

    switch(strategy.item<int32_t>())  // how to read tensor's content using the C++ API: https://stackoverflow.com/a/54208912
    {
        case 0:  // expectation
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "logistic_forward_expectation_cuda_kernel",
                ([&] {
                    forward_expectation_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );
            break;

        case 1:  // argmax sampling (i.e., mode)
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "logistic_forward_mode_cuda_kernel",
                ([&] {
                    forward_mode_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );
            break;

        case 2:  // random sampling
            auto us = torch::rand_like(x_in);
            AT_DISPATCH_FLOATING_TYPES(
                x_in.type(),
                "logistic_forward_random_cuda_kernel",
                ([&] {
                    forward_random_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                        x_out.data_ptr<scalar_t>(),
                        us.data_ptr<scalar_t>(),
                        pmf.data_ptr<scalar_t>(),
                        x_in.numel(),
                        q.data_ptr<scalar_t>(),
                        t.numel()
                    );
                })
            );
            break;

    }

    return x_out;
}


torch::Tensor logistic_backward_cuda_dispatch(
    torch::Tensor grad_in,
    torch::Tensor x_in,
    torch::Tensor q,
    torch::Tensor t,
    torch::Tensor mi,
    torch::Tensor sigma
)
{
    auto grad_out = torch::zeros_like(x_in);
    const dim3 blocks((x_in.numel() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "logistic_backward_cuda",
        ([&] {
            logistic_backward_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                grad_out.data_ptr<scalar_t>(),
                grad_in.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                x_in.numel(),
                q.data_ptr<scalar_t>(),
                t.data_ptr<scalar_t>(),
                t.numel(),
                mi.data_ptr<scalar_t>(),
                sigma.data_ptr<scalar_t>()
            );
        })
    );

    return grad_out;
}
